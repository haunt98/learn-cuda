
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int col = threadIdx.y + blockIdx.y * blockDim.y;
	int half = filterWidth / 2;

	if(row < height && col < width){
		float x = 0;
		float y = 0;
		float z = 0;

		for (int rowF = -half ; rowF <= half; rowF += 1) {
			for (int colF = -half; colF <= half; colF += 1){
				int newRow = row + rowF;
				int newCol = col + colF;
	
				if (newRow < 0) {
					newRow = 0;
				}
				
				if (newRow >= height) {
					newRow = height - 1;
				}
	
				if (newCol < 0) {
					newCol = 0;
				}
				
				if (newCol >= width) {
					newCol = width - 1;
				}
	
				int cord = newRow * width + newCol;
				int cordF = (rowF + half) * filterWidth + (colF + half);
				
				x += inPixels[cord].x * filter[cordF];
				y += inPixels[cord].y * filter[cordF];
				z += inPixels[cord].z * filter[cordF];
			}
		}
		
		outPixels[row * width + col].x = int(x);
		outPixels[row * width + col].y = int(y);
		outPixels[row * width + col].z = int(z);
	}
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
		int half = filterWidth / 2;

		for(int row = 0; row < height; row += 1) {
			for(int col = 0; col < width; col += 1) {
				float x = 0;
				float y = 0;
				float z = 0;
		
				for (int rowF = -half ; rowF <= half; rowF += 1) {
					for (int colF = -half; colF <= half; colF += 1){
						int newRow = row + rowF;
						int newCol = col + colF;
			
						if (newRow < 0) {
							newRow = 0;
						}
						
						if (newRow >= height) {
							newRow = height - 1;
						}
			
						if (newCol < 0) {
							newCol = 0;
						}
						
						if (newCol >= width) {
							newCol = width - 1;
						}
			
						int cord = newRow * width + newCol;
						int cordF = (rowF + half) * filterWidth + (colF + half);
						
						x += inPixels[cord].x * filter[cordF];
						y += inPixels[cord].y * filter[cordF];
						z += inPixels[cord].z * filter[cordF];
					}
				}
			
				outPixels[row * width + col].x = int(x);
				outPixels[row * width + col].y = int(y);
				outPixels[row * width + col].z = int(z);		
			}
		}
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		// Allocate device memories
		uchar3 *d_inPixels, *d_outPixels;
		float *d_filter;
		CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_outPixels, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_filter, filterWidth * filterWidth * sizeof(float)));

		// Copy data to device memories
        CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_filter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice));

		// Set grid size and call kernel (remember to check kernel error)
        dim3 gridSize((height - 1) / blockSize.x + 1, (width - 1) / blockSize.y + 1);
		blurImgKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);

		// Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, width * height * sizeof(uchar3), hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
		CHECK(hipFree(d_filter));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
